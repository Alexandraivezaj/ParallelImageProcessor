#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>

// Dummy kernel
__global__ void dummyKernel(unsigned char* data) {
    // No-op
}

int main() {
    cv::Mat input = cv::imread("../data/sample.jpg", cv::IMREAD_COLOR);
    if (input.empty()) {
        std::cerr << "Error loading image!" << std::endl;
        return -1;
    }

    cv::Mat gray;
    cv::cvtColor(input, gray, cv::COLOR_BGR2GRAY);

    cv::Mat blurred;
    cv::GaussianBlur(gray, blurred, cv::Size(5, 5), 1.5);

    unsigned char* d_data;
    size_t dataSize = blurred.total() * blurred.elemSize();
    hipMalloc((void**)&d_data, dataSize);
    hipMemcpy(d_data, blurred.data, dataSize, hipMemcpyHostToDevice);
    dummyKernel<<<1, 1>>>(d_data);
    hipMemcpy(blurred.data, d_data, dataSize, hipMemcpyDeviceToHost);
    hipFree(d_data);

    cv::Mat edges;
    cv::Canny(blurred, edges, 100, 200);
    cv::imwrite("../results/cuda_edges.jpg", edges);
    std::cout << "✅ CUDA (stub) output saved to results/cuda_edges.jpg" 
<< std::endl;

    return 0;
}

